#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <stdio.h>
#include <fstream> 
#include <time.h>
#include <string>
#include <sstream>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"

#define TMAX 100
#define length 1500000
#define trials 5
#define Num_IPR 10

using namespace std;

__global__ void puffsim(double Jleak, double IP3, double V24, double k24, double n24, double kn24, double nn24,
	double a24, double V42, double k42, double n42, double kn42, double nn42, double a42, int ah42, int Vh42, int Kh42, int BT, int k_on, int Kd, int k_off, double B_new,
	double h24_inf, double h42_inf, double m24_inf, double m42_inf, double *time, double *B, double *c, double *states, double *h42_track, double *m42_track);

__global__ void puffsim(double Jleak, double IP3, double V24, double k24, double n24, double kn24, double nn24,
	double a24, double V42, double k42, double n42, double kn42, double nn42, double a42, int ah42, int Vh42, int Kh42, int BT, int k_on, int Kd, int k_off, double B_new,
	double h24_inf, double h42_inf, double m24_inf, double m42_inf, double *time, double *B, double *c, double *states, double *h42_track, double *m42_track)
{
	printf("Launch kernel");

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int block = blockIdx.x;

	double dt0 = 0.00010; // default time stepsize(second)

	double Jrelease = 200; // calcium released via a single IP3R channel(nM)

	double c0 = 0.1; // resting calcium concn

					 // IPR parameters

					 // state transition rates

	double q45 = 11.000;
	double q54 = 3330;
	double q12 = 1240;
	double q21 = 88;
	double q23 = 3;
	double q32 = 69;
	double q26 = 10500;
	double q62 = 4010;

	int Vs = 4000;
	int Ks = 12;
	int q24scale = 1;
	int q42scale = 1;
	int lambda_h42_scale = 1;

	int lambda_h24 = 40;
	int lambda_m24 = 100;
	int lambda_m42 = 100;

	double dt = dt0;
	double Y0_1;
	double Y0_2;
	double Jserca;
	double YK1_1;
	double YK1_2;
	double Y1_1;
	double Y1_2;
	double YK2_1;
	double YK2_2;
	double Y2_1;
	double Y2_2;
	double YK3_1;
	double YK3_2;
	double Y3_1;
	double Y3_2;
	double YK4_1;
	double YK4_2;
	double Y_new_1;
	double Y_new_2;
	double c_new;
	//double B_new;
	double YK1;
	double Y1;
	double YK2;
	double Y2;
	double YK3;
	double Y3;
	double YK4;
	double q24;
	double q42;
	double lambda_h42;

	double dt1[Num_IPR];

	double r1[Num_IPR];
	double cm[Num_IPR];
	double g[Num_IPR];
	double h24[Num_IPR];
	double h24_new[Num_IPR];
	double m24[Num_IPR];
	double m24_new[Num_IPR];
	double h42[Num_IPR];
	double h42_new[Num_IPR];
	double m42[Num_IPR];
	double m42_new[Num_IPR];


	double Ta_old[36];
	double Ta_new[36];
	for (int x = 0; x < 36; x++) {
		Ta_old[x] = 0;
		Ta_new[x] = 0;
	}

	double dtmin = dt0;
	int index = 0;

	double previous[6];
	int current[10];
	int count = 0;

	// Initial values
	c[idx * length + count] = c0;
	B[idx * length + count] = k_off * BT / (k_on*c0 + k_off);
	time[idx * length + count] = 0;
	int state[Num_IPR];
	for (int w = 0; w < Num_IPR; w++) {
		state[w] = 4;
	}
	
	hiprandState rndState;
	hiprand_init(clock64(), 13, 0, &rndState);
	for (int n = 0; n < Num_IPR; n++)
	{
		h24[n] = h24_inf;
		h24_new[n] = h24[n];
		m24[n] = m24_inf;
		m24_new[n] = m24[n];
		m42[n] = m42_inf;
		m42_new[n] = m42[n];
		h42[n] = h42_inf;
		h42_new[n] = h42[n];
		g[n] = 0;
		cm[n] = c0;
		r1[n] = hiprand_uniform(&rndState);
	}

	int hitpost = 0;
	
	while (time[idx * length + count] < TMAX) {
		/*int rr = idx * length + count;
		if (block == 1) {
			printf("index: %d\n", rr);
			double tm = time[idx * length + count];
			printf("time: %lf\n" ,tm);
		}*/
		
		int No = 0;
		for (int openind = 0; openind < Num_IPR; openind++) {
			if (state[openind] == 5 || state[openind] == 6) {
				No++;
			}
		}
		Y0_1 = c[idx * length + count];
		Y0_2 = B[idx * length + count];

		// Using Runge Kutta 4th order to solve ODEs for [Ca2+] and [fluo4]
		Jserca = Vs * Y0_1 / (Y0_1 + Ks);

		YK1_1 = dt * (Jrelease * No + Jleak - Jserca + (k_off*(BT - B[idx * length + count])) - (k_on* c[idx * length + count] * B[idx * length + count]));
		YK1_2 = dt * (k_off*(BT - B[idx * length + count]) - k_on * c[idx * length + count] * B[idx * length + count]);
		Y1_1 = Y0_1 + YK1_1 / 2;
		Y1_2 = Y0_2 + YK1_2 / 2;
		Jserca = Vs * Y1_1 / (Y1_1 + Ks);

		YK2_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y1_2) - k_on * Y1_1 * Y1_2);
		YK2_2 = dt * (k_off*(BT - Y1_2) - k_on * Y1_1 * Y1_2);
		Y2_1 = Y0_1 + YK2_1 / 2;
		Y2_2 = Y0_2 + YK2_2 / 2;
		Jserca = Vs * Y2_1 / (Y2_1 + Ks);

		YK3_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y2_2) - k_on * Y2_1 * Y2_2);
		YK3_2 = dt * (k_off*(BT - Y2_2) - k_on * Y2_1 * Y2_2);
		Y3_1 = Y0_1 + YK3_1;
		Y3_2 = Y0_2 + YK3_2;
		Jserca = Vs * Y3_1 / (Y3_1 + Ks);

		YK4_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y3_2) - k_on * Y3_1 * Y3_2);
		YK4_2 = dt * (k_off*(BT - Y3_2) - k_on * Y3_1 * Y3_2);

		Y_new_1 = Y0_1 + (YK1_1 + 2 * YK2_1 + 2 * YK3_1 + YK4_1) / 6;
		Y_new_2 = Y0_2 + (YK1_2 + 2 * YK2_2 + 2 * YK3_2 + YK4_2) / 6;
		c_new = Y_new_1; 		
		B_new = Y_new_2;


		for (int i = 0; i < Num_IPR; i++) {
			dt1[i] = dt;
		}

		for (int i = 0; i < Num_IPR; i++) {

			//in a closed state
			if (state[i] - 4.5 < 0) {
				cm[i] = c_new;
			}
			else if (state[i] - 4.5 > 0) {
				cm[i] = c_new + 120; // open state
			}
			else {
				cm[i] = c_new + 60;
			}

			// Updating transition rates

			m42_inf = powf(cm[i], n42) / (powf(k42, n42) + powf(cm[i], n42));
			YK1 = dt * lambda_m42*(m42_inf - m42[i]);
			Y1 = m42[i] + YK1 / 2;
			YK2 = dt * lambda_m42*(m42_inf - Y1);
			Y2 = m42[i] + YK2 / 2;
			YK3 = dt * lambda_m42*(m42_inf - Y2);
			Y3 = m42[i] + YK3;
			YK4 = dt * lambda_m42*(m42_inf - Y3);
			m42_new[i] = m42[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;

			h42_inf = powf(kn42, nn42) / (powf(kn42, nn42) + powf(cm[i], nn42));
			lambda_h42 = lambda_h42_scale * (ah42 + Vh42 * powf(cm[i], 7) / (powf(Kh42, 7) + powf(cm[i], 7)));
			YK1 = dt * lambda_h42 * (h42_inf - h42[i]);
			Y1 = h42[i] + YK1 / 2;
			YK2 = dt * lambda_h42 * (h42_inf - Y1);
			Y2 = h42[i] + YK2 / 2;
			YK3 = dt * lambda_h42 * (h42_inf - Y2);
			Y3 = h42[i] + YK3;
			YK4 = dt * lambda_h42 * (h42_inf - Y3);
			h42_new[i] = h42[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;

			m24_inf = powf(cm[i], n24) / (powf(k24, n24) + powf(cm[i], n24));
			YK1 = dt * lambda_m24*(m24_inf - m24[i]);
			Y1 = m24[i] + YK1 / 2;
			YK2 = dt * lambda_m24*(m24_inf - Y1);
			Y2 = m24[i] + YK2 / 2;
			YK3 = dt * lambda_m24*(m24_inf - Y2);
			Y3 = m24[i] + YK3;
			YK4 = dt * lambda_m24*(m24_inf - Y3);
			m24_new[i] = m24[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;

			h24_inf = powf(kn24, nn24) / (powf(kn24, nn24) + powf(cm[i], nn24));
			YK1 = dt * lambda_h24*(h24_inf - h24[i]);
			Y1 = h24[i] + YK1 / 2;
			YK2 = dt * lambda_h24*(h24_inf - Y1);
			Y2 = h24[i] + YK2 / 2;
			YK3 = dt * lambda_h24*(h24_inf - Y2);
			Y3 = h24[i] + YK3;
			YK4 = dt * lambda_h24*(h24_inf - Y3);
			h24_new[i] = h24[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;

			q24 = q24scale * (a24 + V24 * (1 - m24[i] * h24[i]));
			q42 = q42scale * (a42 + V42 * m42[i] * h42[i]);


			Ta_old[1] = q21;
			Ta_old[6] = q12;
			Ta_old[8] = q32;
			Ta_old[9] = q42;
			Ta_old[11] = q62;
			Ta_old[13] = q23;
			Ta_old[19] = q24;
			Ta_old[22] = q54;
			Ta_old[27] = q45;
			Ta_old[31] = q26;

			q24 = q24scale * (a24 + V24 * (1 - m24_new[i] * h24_new[i]));
			q42 = q42scale * (a42 + V42 * m42_new[i] * h42_new[i]);

			Ta_new[1] = q21;
			Ta_new[6] = q12;
			Ta_new[8] = q32;
			Ta_new[9] = q42;
			Ta_new[11] = q62;
			Ta_new[13] = q23;
			Ta_new[19] = q24;
			Ta_new[22] = q54;
			Ta_new[27] = q45;
			Ta_new[31] = q26;

			double g_old = g[i];
			double sum1 = 0;
			double sum2 = 0;
			int p = state[i] - 1;
			for (int x = 0; x < 6; ++x) {
				sum1 += Ta_old[x * 6 + p];
			}
			for (int y = 0; y < 6; ++y) {
				sum2 += Ta_new[y * 6 + p];
			}
			double g_new = g_old + (sum1 + sum2) / 2 * dt; 
			double epsilon = log(1 / r1[i]); //The threshold for an event to occur
			if (g_new >= epsilon) 
			{ 
				dt1[i] = (epsilon - g_old) / (g_new - g_old)*dt; // changing timestep based on threshold
			}
		}
		for (int j = 0; j < Num_IPR; ++j) {
			dt1[j] = abs(dt1[j]);
		}
		dtmin = dt1[0];
		index = 0;
		// finding minimum time step from receptors
		for (int j = 0; j < Num_IPR; ++j) {
			if (dt1[j] < dtmin) {
				dtmin = dt1[j];
				index = j;
			}
		}
		if (index == 0 && (dtmin / dt) == 1) {
			index = 100;
		}
		else {
			hitpost = 1;
		}
		dt = dtmin; // assigning a new time step value based on IP3 receptor that gave lowest time step

					// Runge Kutta 4th Order to solve ODE using the updated time step

		Jserca = Vs * Y0_1 / (Y0_1 + Ks);

		YK1_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - B[idx * length + count]) - k_on * c[idx * length + count] * B[idx * length + count]);
		YK1_2 = dt * (k_off*(BT - B[idx * length + count]) - k_on * c[idx * length + count] * B[idx * length + count]);
		Y1_1 = Y0_1 + YK1_1 / 2;
		Y1_2 = Y0_2 + YK1_2 / 2;
		Jserca = Vs * Y1_1 / (Y1_1 + Ks);

		YK2_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y1_2) - k_on * Y1_1 * Y1_2);
		YK2_2 = dt * (k_off*(BT - Y1_2) - k_on * Y1_1 * Y1_2);
		Y2_1 = Y0_1 + YK2_1 / 2;
		Y2_2 = Y0_2 + YK2_2 / 2;
		Jserca = Vs * Y2_1 / (Y2_1 + Ks);

		YK3_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y2_2) - k_on * Y2_1 * Y2_2);
		YK3_2 = dt * (k_off*(BT - Y2_2) - k_on * Y2_1 * Y2_2);
		Y3_1 = Y0_1 + YK3_1;
		Y3_2 = Y0_2 + YK3_2;
		Jserca = Vs * Y3_1 / (Y3_1 + Ks);

		YK4_1 = dt * (Jrelease*No + Jleak - Jserca + k_off * (BT - Y3_2) - k_on * Y3_1 * Y3_2);
		YK4_2 = dt * (k_off*(BT - Y3_2) - k_on * Y3_1 * Y3_2);

		Y_new_1 = Y0_1 + (YK1_1 + 2 * YK2_1 + 2 * YK3_1 + YK4_1) / 6;
		Y_new_2 = Y0_2 + (YK1_2 + 2 * YK2_2 + 2 * YK3_2 + YK4_2) / 6;
		c_new = Y_new_1;
		B_new = Y_new_2;

		//substitute for heaviside function
		for (int i = 0;i < Num_IPR;i++)
		{
			if (state[i] - 4.5 < 0)
			{
				cm[i] = c_new; 
			}
			else if (state[i] - 4.5 > 0)
			{
				cm[i] = c_new + 120; 
			}
			else if (state[i] - 4.5 == 0)
			{
				cm[i] = c_new + 60; 
			}

			// updating transition states
			m42_inf = powf(cm[i], n42) / (powf(k42, n42) + powf(cm[i], n42));
			YK1 = dt * lambda_m42*(m42_inf - m42[i]);
			Y1 = m42[i] + YK1 / 2;
			YK2 = dt * lambda_m42*(m42_inf - Y1);
			Y2 = m42[i] + YK2 / 2;
			YK3 = dt * lambda_m42*(m42_inf - Y2);
			Y3 = m42[i] + YK3;
			YK4 = dt * lambda_m42*(m42_inf - Y3);
			m42_new[i] = m42[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;
			h42_inf = powf(kn42, nn42) / (powf(kn42, nn42) + powf(cm[i], nn42));
			double tau_h42 = ah42 + Vh42 * powf(cm[i], 7) / (powf(Kh42, 7) + powf(cm[i], 7));
			YK1 = dt * tau_h42*(h42_inf - h42[i]);
			Y1 = h42[i] + YK1 / 2;
			YK2 = dt * tau_h42*(h42_inf - Y1);
			Y2 = h42[i] + YK2 / 2;
			YK3 = dt * tau_h42*(h42_inf - Y2);
			Y3 = h42[i] + YK3;
			YK4 = dt * tau_h42*(h42_inf - Y3);
			h42_new[i] = h42[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;
			m24_inf = powf(cm[i], n24) / (powf(k24, n24) + powf(cm[i], n24));
			YK1 = dt * lambda_m24*(m24_inf - m24[i]);
			Y1 = m24[i] + YK1 / 2;
			YK2 = dt * lambda_m24*(m24_inf - Y1);
			Y2 = m24[i] + YK2 / 2;
			YK3 = dt * lambda_m24*(m24_inf - Y2);
			Y3 = m24[i] + YK3;
			YK4 = dt * lambda_m24*(m24_inf - Y3);
			m24_new[i] = m24[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;
			h24_inf = powf(kn24, nn24) / (powf(kn24, nn24) + powf(cm[i], nn24));
			YK1 = dt * lambda_h24*(h24_inf - h24[i]);
			Y1 = h24[i] + YK1 / 2;
			YK2 = dt * lambda_h24*(h24_inf - Y1);
			Y2 = h24[i] + YK2 / 2;
			YK3 = dt * lambda_h24*(h24_inf - Y2);
			Y3 = h24[i] + YK3;
			YK4 = dt * lambda_h24*(h24_inf - Y3);
			h24_new[i] = h24[i] + (YK1 + 2 * YK2 + 2 * YK3 + YK4) / 6;

			q24 = q24scale * (a24 + V24 * (1 - m24[i] * h24[i]));
			q42 = q42scale * (a42 + V42 * m42[i] * h42[i]);

			Ta_old[1] = q21;
			Ta_old[6] = q12;
			Ta_old[8] = q32;
			Ta_old[9] = q42;
			Ta_old[11] = q62;
			Ta_old[13] = q23;
			Ta_old[19] = q24;
			Ta_old[22] = q54;
			Ta_old[27] = q45;
			Ta_old[31] = q26;

			q24 = q24scale * (a24 + V24 * (1 - m24_new[i] * h24_new[i]));
			q42 = q42scale * (a42 + V42 * m42_new[i] * h42_new[i]);

			Ta_new[1] = q21;
			Ta_new[6] = q12;
			Ta_new[8] = q32;
			Ta_new[9] = q42;
			Ta_new[11] = q62;
			Ta_new[13] = q23;
			Ta_new[19] = q24;
			Ta_new[22] = q54;
			Ta_new[27] = q45;
			Ta_new[31] = q26;
			int p = state[i] - 1; //p is row
			//int check = state[i];
			double sum5 = Ta_old[p];
			double sum6 = Ta_new[p];
			for (int j = 1; j < 6; j++) //j is column 0-5 i.e. all columns
			{
				sum5 = sum5 + Ta_old[j * 6 + p];
			}
			for (int j = 1; j < 6; j++)
			{
				sum6 = sum6 + Ta_new[j * 6 + p];
			}
			g[i] = g[i] + (sum5 + sum6) / 2 * dt;

		}
		
		//updating the loop
		c[idx * length + count + 1] = c_new;
		B[idx * length + count + 1] = B_new;
		//printf("B: %lf\n", B[idx * length + count]);
		time[idx * length + count + 1] = time[idx * length + count] + dt;
		count++;
		
		//updating track variables
		for (int q = 0; q < Num_IPR; q++)
		{
			h24[q] = h24_new[q];
			m24[q] = m24_new[q];
			h42[q] = h42_new[q];
			m42[q] = m42_new[q];
			h42_track[count * Num_IPR + q] = h42_new[q];
			m42_track[count  *Num_IPR + q] = m42_new[q];
		}

		if (index != 100) {
			q24 = q24scale * (a24 + V24 * (1 - m24[index] * h24[index]));
			q42 = q42scale * (a42 + V42 * m42[index] * h42[index]);

			Ta_new[1] = q12;
			Ta_new[6] = q21;
			Ta_new[8] = q23;
			Ta_new[9] = q24;
			Ta_new[11] = q26;
			Ta_new[13] = q32;
			Ta_new[19] = q42;
			Ta_new[22] = q45;
			Ta_new[27] = q54;
			Ta_new[31] = q62;

			double r2 = hiprand_uniform(&rndState);
			for (int k = 0; k <= 5; k++) 
			{
				int p = state[index] - 1;  
				int var = 6 * p + k;
				previous[k] = Ta_new[var]; 
			}
			int compare = 0;
			double sum1 = 0;
			double sum2 = 1;
			while ((sum1 / sum2) < r2)
			{
				sum1 = 0;
				sum2 = 0;
				for (int j = 0; j <= compare; j++)
				{
					sum1 += previous[j]; //The sum of the first 1, 2, ..., 6 elements of previous[]
				}
				for (int h = 0; h < 6; h++)
				{
					sum2 += previous[h]; //The sum of all 6 elements in previous[]
				}
				compare++;
				
				if (compare >= 6)
				{
					break;
				}
			}
			for (int y = 0; y < Num_IPR; y++)
			{
				current[y] = state[y];
			}
			current[index] = compare; 
			for (int y = 0; y < Num_IPR; y++)
			{
				//printf("state[y]: %lf\n", state[y]);
				state[y] = current[y]; 
				
			}
			r1[index] = hiprand_uniform(&rndState); //Assign new random number to r1 for the receptor that just changes its state
			g[index] = 0;
		}
		else {

			for (int y = 0; y < Num_IPR; y++)
			{
				state[y] = state[y]; //state does not change
			}
			dt = dt0;
		}
		
		for (int t = 0; t < Num_IPR; t++) {
			states[idx * Num_IPR * count + t] = state[t];
		}
	}

	printf("end kernel\n");
}

int main()
{

	double dt0 = 0.0001; // default time stepsize(second)

	double Jrelease = 200; // calcium released via a single IP3R channel(nM)

	double c0 = 0.1; // resting calcium concn

	//IPR parameters

		// state transition rates

	double q45 = 11.000;
	double q54 = 3330;
	double q12 = 1240;
	double q21 = 88;
	double q23 = 3;
	double q32 = 69;
	double q26 = 10500;
	double q62 = 4010;


	int Vs = 4000;
	int Ks = 12;

	double Jleak = Vs * 0.1 / (0.1 + Ks); // = 33 muM / s

	double IP3 = 0.1; // muM

	double V24 = 100;
	double k24 = 0.5490;
	double n24 = 6.3119;
	double kn24 = 96.9114;
	double nn24 = 0.0363;
	double a24 = 1 + 7.5 / (pow(IP3, 2) + 0.25);
	double V42 = 100;
	double k42 = 0.4;
	double n42 = 11.1414;
	double kn42 = 0.1703;
	double nn42 = 3.2287;
	double a42 = 1.8 * pow(IP3, 2) / (pow(IP3, 2) + 0.34);


	int ah42 = 1;
	int Vh42 = 100;
	int Kh42 = 20;

	//dye buffer
	int BT = 20; // total calmodulin buffer
	int k_on = 150; // on - rate or binding rate
	int Kd = 2; // dissociation constant (muM)
	int k_off = Kd * k_on; // dissociation rate
	double B_var = k_off * BT / (k_on * 0.1 + k_off); // resting free buffer concn
	double B_new = B_var;

	double h24_inf = pow(kn24, nn24) / (pow(kn24, nn24) + pow(c0, nn24));
	double m24_inf = pow(c0, n24) / (pow(k24, n24) + pow(c0, n24));
	double m42_inf = pow(c0, n42) / (pow(k42, n42) + pow(c0, n42));
	double h42_inf = pow(kn42, nn42) / (pow(kn42, nn42) + pow(c0, nn42));

	// sending memory to CUDA kernel

	
	

	double *host_B = new double[trials*length];
	double *B = new double[trials * length];
	hipMalloc((void**)&B, trials *length * sizeof(double));
	hipMemcpy(B, host_B, trials * length * sizeof(double), hipMemcpyHostToDevice);


	double *host_c = new double[trials * length];
	double *host_time = new double[trials*length];
	double *time = new double[trials*length];

	for (int i = 0; i < (trials * length); i++) {
		host_c[i] = 0;
		host_time[i] = 0;
		host_B[i] = 0;
	}
	if (hipSuccess != hipMalloc((void**)&time, trials * length * sizeof(double))) {
		cout << "malloc fail" << endl;
	}

	if (hipSuccess != hipMemcpy(time, host_time, trials *  length * sizeof(double), hipMemcpyHostToDevice)) {
		cout << "memcpy fail #1" << endl;
	}

	double *c = new double[trials*length];
	hipMalloc((void**)&c, trials * length * sizeof(double));
	hipMemcpy(c, host_c, trials*length * sizeof(double), hipMemcpyHostToDevice);


	double *host_states = new double[trials * Num_IPR * length];
	double *states = new double[trials * Num_IPR * length];
	hipMalloc((void**)&states,trials * Num_IPR * length * sizeof(double));
	hipMemcpy(states, host_states, trials *Num_IPR * length * sizeof(double), hipMemcpyHostToDevice);
	

	/*double *host_g = new double[trials * Num_IPR];
	double *g = new double[trials * Num_IPR];
	hipMalloc((void**)&g, trials * Num_IPR * sizeof(double));
	hipMemcpy(g, host_g, trials * Num_IPR * sizeof(double), hipMemcpyHostToDevice); */


	/*double *host_cm = new double[Num_IPR];
	for (int i = 0; i < Num_IPR; i++) {
		host_cm[i] = 0.1;
 	}
	double *cm = new double[Num_IPR];
	hipMalloc((void**)&cm, Num_IPR * sizeof(double));
	hipMemcpy(cm, host_cm, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_r1 = new double[Num_IPR];
	double *r1 = new double[Num_IPR];
	hipMalloc((void**)&r1, Num_IPR * sizeof(double));
	hipMemcpy(r1, host_r1, Num_IPR * sizeof(double), hipMemcpyHostToDevice); */

	double *host_h24 = new double[Num_IPR];
	for (int k = 0; k < Num_IPR; k++) {
		host_h24[k] = h24_inf;
	}
	double *h24 = new double[Num_IPR];
	hipMalloc((void**)&h24, Num_IPR * sizeof(double));
	hipMemcpy(h24, host_h24, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_h24_new = new double[Num_IPR];
	for (int i = 0; i < Num_IPR; i++) {
		host_h24_new[i] = host_h24[i];
	}
	double *h24_new = new double[Num_IPR];
	hipMalloc((void**)&h24_new, Num_IPR * sizeof(double));
	hipMemcpy(h24_new, host_h24_new, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_m24 = new double[Num_IPR];
	for (int k = 0; k < Num_IPR; k++) {
		host_m24[k] = m24_inf;
	}
	double *m24 = new double[Num_IPR];
	hipMalloc((void**)&m24, Num_IPR * sizeof(double));
	hipMemcpy(m24, host_m24, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_m24_new = new double[Num_IPR];
	for (int i = 0; i < Num_IPR; i++) {
		host_m24_new[i] = host_m24[i];
	}
	double *m24_new = new double[Num_IPR];
	hipMalloc((void**)&m24_new, Num_IPR * sizeof(double));
	hipMemcpy(m24_new, host_m24_new, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_h42 = new double[Num_IPR];
	for (int k = 0; k < Num_IPR; k++) {
		host_h42[k] = h42_inf;
	}
	double *h42 = new double[Num_IPR];
	hipMalloc((void**)&h42, Num_IPR * sizeof(double));
	hipMemcpy(h42, host_h42, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_h42_new = new double[Num_IPR];
	for (int i = 0; i < Num_IPR; i++) {
		host_h42_new[i] = host_h42[i];
	}
	double *h42_new = new double[Num_IPR];
	hipMalloc((void**)&h42_new, Num_IPR * sizeof(double));
	hipMemcpy(h42_new, host_h42_new, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_h42_track = new double[Num_IPR * length];
	double *h42_track = new double[Num_IPR * length];
	hipMalloc((void**)&h42_track, Num_IPR * length * sizeof(double));

	double *host_m42 = new double[Num_IPR];
	for (int k = 0; k < Num_IPR; k++) {
		host_m42[k] = m42_inf;
	}
	double *m42 = new double[Num_IPR];
	hipMalloc((void**)&m42, Num_IPR * sizeof(double));
	hipMemcpy(m42, host_m42, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_m42_new = new double[Num_IPR];
	for (int i = 0; i < Num_IPR; i++) {
		host_m42_new[i] = host_m42[i];
	}
	double *m42_new = new double[Num_IPR];
	hipMalloc((void**)&m42_new, Num_IPR * sizeof(double));
	hipMemcpy(m42_new, host_m42_new, Num_IPR * sizeof(double), hipMemcpyHostToDevice);

	double *host_m42_track = new double[Num_IPR * length];
	double *m42_track = new double[Num_IPR * length];
	hipMalloc((void**)&m42_track, Num_IPR * length * sizeof(double));
	
	
	for (int i = 0; i < (Num_IPR); i++) {
		host_h42_track[i] = host_h42[i];
		host_m42_track[i] = host_m42[i];
	}
	hipMemcpy(m42_track, host_m42_track, Num_IPR * length * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h42_track, host_h42_track, Num_IPR * length * sizeof(double), hipMemcpyHostToDevice);

	puffsim << <trials, 1 >> > (Jleak, IP3, V24, k24, n24, kn24, nn24, a24, V42, k42, n42, kn42, nn42, a42, ah42, Vh42, Kh42, BT, k_on, Kd, k_off, B_new,
		h24_inf, h42_inf, m24_inf, m42_inf, time, B, c, states,  h42_track, m42_track);
	hipDeviceSynchronize();

	//hipMemcpy(host_time, time, trials * length * sizeof(double), hipMemcpyDeviceToHost);
	if (hipSuccess != hipMemcpy(host_time, time, trials * length * sizeof(double), hipMemcpyDeviceToHost)) {
		cout << "memcpy fail" << endl;
	}
	if (hipSuccess != hipMemcpy(host_c, c, trials * length * sizeof(double), hipMemcpyDeviceToHost)) { 
		cout << "memcpy fail calcium" << endl;
	}
	if (hipSuccess != hipMemcpy(host_B, B, trials * length * sizeof(double), hipMemcpyDeviceToHost)) {
		cout << "memcpy fail B" << endl;
	}

	hipFree(time);
	hipFree(c);
	hipFree(B);

	double **Calcium = new double *[length];
	for (int i = 0; i < length; i++) {
		Calcium[i] = new double[trials];
	}
	double **TIME = new double *[length];
	for (int i = 0; i < length; i++) {
		TIME[i] = new double[trials];
	}

	double **B_arr = new double *[length];
	for (int i = 0; i < length; i++) {
		B_arr[i] = new double[trials];
	}
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < trials; j++) {
			Calcium[i][j] = 0.0000000000000;
			TIME[i][j] = 0.00000000000000;
			B_arr[i][j] = 0.00000000000000;

		}
	}


	
	int count = 0;
	for (int x = 0; x < trials; x++) {
		for (int y = 0; y < length; y++) {
			//cout << "x: " << x << endl;
			//cout << "y: " << y << endl;
			//cout << "hosttime: " << host_time[count] << endl;
			//cout << "count: " << count << endl;
			//cout << "Calcium: " << Calcium[x][y] << endl;
			
			Calcium[y][x] = host_c[count];
			TIME[y][x] = host_time[count];
			B_arr[y][x] = host_B[count];
			count++;
		}
	}

	ofstream cal;
	cal.open("Calcium.csv");
	while (cal.is_open()) {
		for (int i = 0; i < length; i++) {
			cal << Calcium[i][0];
			for (int j = 1; j < trials; j++) {
				cal << "," << Calcium[i][j];
			}
			cal << endl;
		}
		break;
	}
	cal.close();

	ofstream plottime;
	plottime.open("Time.csv");
	while (plottime.is_open()) {
		for (int i = 0; i < length; i++) {
			plottime << TIME[i][0];
			for (int j = 1; j < trials; j++) {
				plottime << "," << TIME[i][j];
			}
			plottime << endl;
		}
		break;
	}
	plottime.close();

	ofstream Bfile;
	Bfile.open("B.csv");
	while (Bfile.is_open()) {
		for (int i = 0; i < length; i++) {
			Bfile << B_arr[i][0];
			for (int j = 1; j < trials; j++) {
				Bfile << "," << B_arr[i][j];
			}
			Bfile << endl;
		}
		break;
	}
	Bfile.close();

	printf("Program finished.");
	
		//USE CUDAFREE()
	// delete pointers
	delete[] host_time;
	delete[] host_c;
	delete[] host_B;
	hipFree(states);
	hipFree(h42_track);
	delete[] host_h42_track;
	hipFree(m42_track);
	delete[] host_m42_track; 

}